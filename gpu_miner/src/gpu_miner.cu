#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>
#include <hip/hip_runtime.h>

// TODO: Implement function to search for all nonces from 1 through MAX_NONCE (inclusive) using CUDA Threads
__global__ void findNonce(BYTE *d_block_content, int current_length, BYTE *block_hash, uint64_t *index, BYTE *dif,int* check) {
	uint64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	BYTE d_block_content_copy[BLOCK_SIZE];
	d_strcpy((char*)d_block_content_copy,(char*) d_block_content);
	BYTE d_block_hash[SHA256_HASH_SIZE];
	uint64_t nonce = thread_id;
	char nonce_string[NONCE_SIZE];
	intToString(nonce, nonce_string);
	d_strcpy((char*) d_block_content_copy + current_length, nonce_string);
	if (*check == 0) {
		return;
	}
	apply_sha256(d_block_content_copy, d_strlen((const char*)d_block_content_copy),d_block_hash, *check);
	if (compare_hashes(d_block_hash, dif) <= 0) {
			atomicExch(check, 0);
			*index = nonce;
			memcpy(block_hash, d_block_hash, SHA256_HASH_SIZE);
		}
}

void getParam(int& numBlocks, int& numThreads, int numItems) {
    hipDeviceProp_t prop;
    hipError_t ret;
	ret = hipGetDeviceProperties(&prop, 0);

    if (ret != hipSuccess) {
        printf("hipGetDeviceProperties failed:");
        exit(1);
    }
    numThreads = prop.maxThreadsPerBlock/2;
    numBlocks = numItems / numThreads;
	if (numItems % numThreads) 
		++numBlocks;
}

int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];

	int numBlocks, numThreads;
	hipDeviceProp_t prop;
    hipError_t ret;
	ret = hipGetDeviceProperties(&prop, 0);

    if (ret != hipSuccess) {
        printf("hipGetDeviceProperties failed:");
        exit(1);
    }
    numThreads = prop.maxThreadsPerBlock/2;
    numBlocks = int(MAX_NONCE) / numThreads;
	if (int(MAX_NONCE) % numThreads) 
		++numBlocks;

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	// prev_block_hash + top_hash
	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);
	size_t current_length;
	current_length = strlen((char*) block_content);

	hipEvent_t start, stop;
	BYTE *d_block_content;
	BYTE *d_block_hash;
	BYTE *difficulty;
	uint64_t nonce = 0;
	uint64_t *index;
	int *check;
	hipMallocManaged(&check, sizeof(int));
	*check = 1;
	hipMallocManaged(&index, sizeof(uint64_t));
	hipMallocManaged(&d_block_hash, SHA256_HASH_SIZE * sizeof(BYTE));
	hipMalloc((void**)&difficulty, SHA256_HASH_SIZE);
	hipMalloc((void**)&d_block_content, BLOCK_SIZE);
	hipMemcpy(d_block_content, block_content, BLOCK_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(difficulty, DIFFICULTY, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
	startTiming(&start, &stop);
	findNonce<<<numBlocks,numThreads>>>(d_block_content, current_length,d_block_hash,index,difficulty,check);
	float seconds = stopTiming(&start, &stop);
	FILE *f = fopen("results.csv", "w+");
	if (f == NULL) {
		printf("Error opening file!\n");
		return 1;
	}
	nonce = *index;
	printResult(d_block_hash, nonce, seconds);
	hipFree(d_block_content);
	hipFree(d_block_hash);
	hipFree(difficulty);
	hipFree(index);
	hipFree(check);
	return 0;
}
